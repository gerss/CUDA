
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

double get_time()
{  struct timeval tim;
   gettimeofday(&tim, NULL);
   return (double) tim.tv_sec+(tim.tv_usec/1000000.0);
}

//KERNEL
__global__ void Add(int *a, int *b)
{
a[0] += b[0];
}

int main()
{

//ALLOCATE AND INITIALIZA DATA ON CPU
int a = 5, b = 9;
int *d_a, *d_b;

//ALLOCATE DATA ON GPU
hipMalloc(&d_a, sizeof(int));
hipMalloc(&d_b, sizeof(int));

//TRANSFER DATA FROM CPU TO GPU
hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

double start = get_time(); //Initial time
//RUN KERNEL
Add<<<1,1>>>(d_a, d_b);
double finish = get_time(); //Final time

double diff = finish - start;
cout<<"time ="<<diff<<endl;

//TRANSFER DATA FROM GPU TO CPU
hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

cout<<"Total time is : "<<a<<endl;

//FREE MEMORY
hipFree(d_a);
hipFree(d_b);

return 0;
}
