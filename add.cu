
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

//KERNEL
__global__ void Add(int *a, int *b)
{
a[0] += b[0];
}

int main()
{

//ALLOCATE AND INITIALIZA DATA ON CPU
int a = 5, b = 9;
int *d_a, *d_b;

//ALLOCATE DATA ON GPU
hipMalloc(&d_a, sizeof(int));
hipMalloc(&d_b, sizeof(int));

//TRANSFER DATA FROM CPU TO GPU
hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

//RUN KERNEL
Add<<<1,1>>>(d_a, d_b);

//TRANSFER DATA FROM GPU TO CPU
hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

cout<<"The answer is "<<a<<endl;

hipFree(d_a);
hipFree(d_b);

return 0;
}
