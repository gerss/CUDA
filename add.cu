
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>

using namespace std;

double get_time()
{  struct timeval tim;
   gettimeofday(&tim, NULL);
   return (double) tim.tv_sec+(tim.tv_usec/1000000.0);
}

//KERNEL
__global__ void Add(float *a, float *b, float *c, int N, int BSZ)
{
	int i = blockIdx.x*BSZ + threadIdx.x;

	if(i<N)
	c[i] = a[i] + b[i];
}

int main()
{

//ALLOCATE AND INITIALIZE DATA ON CPU

printf("----------------------------------------\n");
printf(" ALLOCATE AND INITIALIZE DATA ON CPU\n");
printf("----------------------------------------\n");
int N =8388608; //n° of threads
cout<<"N° of threads="<<N<<endl;
int blocksize = 256;
cout<<"N° of threads per block="<<blocksize<<endl;
int num_block = (N-0.5)/blocksize + 1;
cout<<"N° of blocks="<<num_block<<endl;

//float *a, *b, *c;
//a = (float *)malloc(N*sizeof(float));

float *a = new float[N];
float *b = new float[N];
float *c = new float[N];

for( int j=0;j<N;++j)
{
	a[j]=j;
	b[j]=j;
	c[j]=0;
}

cout<<"A[0]="<<a[0]<<endl;
printf(".\n.\n.\n");
cout<<"A[N-1]="<<a[N-1]<<endl;
printf(" + \n");
cout<<"B[0]="<<b[0]<<endl;
printf(".\n.\n.\n");
cout<<"B[N-1]="<<b[N-1]<<endl;

//ALLOCATE DATA ON GPU
printf("----------------------------------------\n");
printf(" ALLOCATE DATA ON GPU\n");
printf("----------------------------------------\n");

float *d_a, *d_b, *d_c;

hipMalloc((void**) &d_a, N*sizeof(float));
hipMalloc((void**) &d_b, N*sizeof(float));
hipMalloc((void**) &d_c, N*sizeof(float));

//cudaMalloc(&d_a, N*sizeof(float));

//TRANSFER DATA FROM CPU TO GPU
printf(" TRANSFER DATA FROM CPU TO GPU\n");
printf("----------------------------------------\n");

hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

//RUN KERNEL
printf(" RUN KERNEL");

double start = get_time(); //Initial time
Add<<<num_block,blocksize>>>(d_a, d_b, d_c, N, blocksize);
double finish = get_time(); //Final time

double diff = finish - start;
cout<<"  time ="<<diff<<" [s]"<<endl;
printf("----------------------------------------\n");

//TRANSFER DATA FROM GPU TO CPU
printf(" TRANSFER DATA FROM GPU TO CPU\n");
printf("----------------------------------------\n");

hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

cout<<"\na[N-1] + b[N-1] = "<<a[N-1]<<" + "<<b[N-1]<<endl;
cout<<"c[N-1]          = "<<c[N-1]<<endl;

//FREE MEMORY
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;
}
