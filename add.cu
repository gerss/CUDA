
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>

using namespace std;

double get_time()
{  struct timeval tim;
   gettimeofday(&tim, NULL);
   return (double) tim.tv_sec+(tim.tv_usec/1000000.0);
}

//KERNEL
__global__ void Add(float *a, float *b, float *c, int N, int BSZ)
{

	int i = blockIdx.x*BSZ + threadIdx.x;

	for(i=0;i<N;i++)
	{
	c[i] = a[i] + b[i];
	}

}

int main()
{

//ALLOCATE AND INITIALIZE DATA ON CPU

printf("\n ALLOCATE AND INITIALIZE DATA ON CPU\n");
printf("----------------------------------------\n");
int N =1048576; //n° of threads
cout<<"N° of threads="<<N<<endl;
int blocksize = 256;
cout<<"N° of threads per block="<<blocksize<<endl;
int num_block = (N-0.5)/blocksize + 1;
cout<<"N° of blocks="<<num_block<<endl;

//float *a, *b, *c;
//a = (float *)malloc(N*sizeof(float));

float *a = new float[N];
float *b = new float[N];
float *c = new float[N];

for( int j=0;j<N;++j)
{
	a[j]=j;
	b[j]=j;
	c[j]=0;
}

printf("\n Vector A\n");
cout<<"A[0]="<<a[0]<<endl;
printf(".\n.\n.\n");
cout<<"A[N-1]="<<a[N-1]<<endl;

printf("\n Vector B\n");
cout<<"B[0]="<<b[0]<<endl;
printf(".\n.\n.\n");
cout<<"B[N-1]="<<b[N-1]<<endl;

//ALLOCATE DATA ON GPU
printf("\n ALLOCATE DATA ON GPU\n");
printf("----------------------------------------");

float *d_a, *d_b, *d_c;

hipMalloc((void**) &d_a, N*sizeof(float));
hipMalloc((void**) &d_b, N*sizeof(float));
hipMalloc((void**) &d_c, N*sizeof(float));

//cudaMalloc(&d_a, N*sizeof(float));

//TRANSFER DATA FROM CPU TO GPU
printf("\n TRANSFER DATA FROM CPU TO GPU\n");
printf("----------------------------------------");

hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_c, c, N*sizeof(float), hipMemcpyHostToDevice);

//RUN KERNEL
printf("\n RUN KERNEL\n");
printf("----------------------------------------");

double start = get_time(); //Initial time
Add<<<num_block,blocksize>>>(d_a, d_b, d_c, N, blocksize);
double finish = get_time(); //Final time

double diff = finish - start;
cout<<"\nTotal time ="<<diff<<" [s]\n"<<endl;

//TRANSFER DATA FROM GPU TO CPU
printf("\n TRANSFER DATA FROM GPU TO CPU\n");
printf("----------------------------------------");

hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

cout<<"\na[N-1] + b[N-1] ="<<a[N-1]<<"+"<<b[N-1]<<endl;
cout<<"C[N-1] = "<<c[N-1]<<endl;

//FREE MEMORY
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;
}
